
#include <hip/hip_runtime.h>
#include <stdio.h>

// kernel to be run on the TX2
__global__ void square(float *d_out, float *d_in){
  int idx = threadIdx.x; // this is how you get the thread index
  if (idx % 3 == 0 || idx % 5 == 0){
    d_out[idx] = idx;
  }
}

// main is here. this is the CPU code. 
int main(){
  // the size of the array, which is really the thread count per block
  const int ARRAY_SIZE = 1000; // max thread count per block on the TX2
  // total bytes in the array
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);

  // generate the input array on the host for the device my dude
  float h_in[ARRAY_SIZE];
  for (int i = 0; i < ARRAY_SIZE; i++){
    h_in[i] = 0;
  }
  float h_out[ARRAY_SIZE]; // make the other array too my guy
  
  // make the pointers that point to the mem locations on the GPU m'lady
  float *d_in;
  float *d_out;

  //allocate the memory on the GPU my man
  hipMalloc((void **) &d_in, ARRAY_BYTES);
  hipMalloc((void **) &d_out, ARRAY_BYTES);

  // transfer the array to the GPU my dude. Copy's contents of h_in to d_in
  hipMemcpy(d_in, h_in, ARRAY_BYTES, hipMemcpyHostToDevice);

  // launch the kernel
  square<<<1,ARRAY_SIZE>>>(d_out,d_in);

  // copy the result back to the CPU mem
  hipMemcpy(h_out, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost);

  // print the result
  int sum = 0;
  for (int i = 0; i < ARRAY_SIZE; i++){
    sum += h_out[i];
  }

  printf("answer: %d",sum);

  // free the device yo
  hipFree(d_in);
  hipFree(d_out);

  return 0;
}
