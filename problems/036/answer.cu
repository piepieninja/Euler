#include "hip/hip_runtime.h"
#include <iostream>
#include <cmath>

using namespace std;

__device int pow(int x, int n){
  while (n != 0){
    x *= x;
    n--;
  }
  return x;
}

// the actual verifier is here
__global__ void double_pal_verify(int *in, int *out, int n){
  // the number we are testing
  int num = blockIdx.x*blockDim.x+threadIdx.x;
  // is it zero? 
  if (!num) { out[0] = 0; return; }
  bool dig;
  if (num < 10) { out[num] = num; return; }
  
  // check if the int is a palindrome
  int temp = num;
  int rmdr, test;
  while (num > 0){
    rmdr = temp % 10;
    test = 10 * test + r;
    temp = temp % 10;
  }
  if (num == test){
    out[num] = num;
    return;
  } else {
    out[num] = 0;
    return;
  }

  // check if the bool is a palindrome
  if (num >= pow()){
    // wow this has 20 digits in binary! 
  } else if () {

  }

}

int main(){
    cout << "PROBLEM 36 SOLUTION" << endl;
    // vector size
    int n = 1000000;

    return 0;
}

